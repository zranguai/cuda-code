// 优化技巧2: 解决共享内存冲突(bank冲突)(最优：线程束中不同线程访问不同存储体)
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>

#define THREAD_PER_BLOCK 256

// idle Threads
__global__ void reduce2(float *d_in,float *d_out){
    __shared__ float sdata[THREAD_PER_BLOCK];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = d_in[i];
    __syncthreads();

    // do reduction in shared mem
    /*
     * 解决共享内存冲突， 共享内存会按线程束进行32等分 以第一次迭代为例：
     * 这里分析0号warp, 0号线程取第0号元素和第128号元素
    */
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) d_out[blockIdx.x] = sdata[0];
}

bool check(float *out,float *res,int n){
    for(int i=0;i<n;i++){
        if(out[i]!=res[i])
            return false;
    }
    return true;
}

int main(){
    const int N=32*1024*1024;
    float *a=(float *)malloc(N*sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a,N*sizeof(float));

    int block_num=N/THREAD_PER_BLOCK;
    float *out=(float *)malloc((N/THREAD_PER_BLOCK)*sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out,(N/THREAD_PER_BLOCK)*sizeof(float));
    float *res=(float *)malloc((N/THREAD_PER_BLOCK)*sizeof(float));

    for(int i=0;i<N;i++){
        a[i]=1;
    }

    for(int i=0;i<block_num;i++){
        float cur=0;
        for(int j=0;j<THREAD_PER_BLOCK;j++){
            cur+=a[i*THREAD_PER_BLOCK+j];
        }
        res[i]=cur;
    }

    hipMemcpy(d_a,a,N*sizeof(float),hipMemcpyHostToDevice);

    dim3 Grid( N/THREAD_PER_BLOCK,1);
    dim3 Block( THREAD_PER_BLOCK,1);

    reduce2<<<Grid,Block>>>(d_a,d_out);

    hipMemcpy(out,d_out,block_num*sizeof(float),hipMemcpyDeviceToHost);

    if(check(out,res,block_num))printf("the ans is right\n");
    else{
        printf("the ans is wrong\n");
        for(int i=0;i<block_num;i++){
            printf("%lf ",out[i]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_out);
}
