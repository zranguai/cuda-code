#include "hip/hip_runtime.h"
/*********************************************************************************************
 * file name  :global_memory.cu
 * brief      : 静态全局变量使用
***********************************************************************************************/

#include <hip/hip_runtime.h>
#include <iostream>
#include "common.cuh"

__device__ int d_x = 1;
__device__ int d_y[2];

__global__ void kernel(void)
{
    d_y[0] += d_x;
    d_y[1] += d_x;

    printf("d_x = %d, d_y[0] = %d, d_y[1] = %d.\n", d_x, d_y[0], d_y[1]);
}



int main(int argc, char **argv)
{
    int devID = 0;
    hipDeviceProp_t deviceProps;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProps, devID));
    std::cout << "运行GPU设备:" << deviceProps.name << std::endl;

    int h_y[2] = {10, 20};
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_y), h_y, sizeof(int) * 2));

    dim3 block(1);
    dim3 grid(1);
    kernel<<<grid, block>>>();
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpyFromSymbol(h_y, HIP_SYMBOL(d_y), sizeof(int) * 2));
    printf("h_y[0] = %d, h_y[1] = %d.\n", h_y[0], h_y[1]);

    CUDA_CHECK(hipDeviceReset());

    return 0;
}