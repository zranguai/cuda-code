#include "hip/hip_runtime.h"
/*********************************************************************************************
 * file name  : global_memory.cu
 * brief      : 常量内存的使用
***********************************************************************************************/

#include <hip/hip_runtime.h>
#include <iostream>
#include "common.cuh"

__constant__ float c_data;
__constant__ float c_data2 = 6.6f;

__global__ void kernel_1(void)
{
    
    printf("Constant data c_data = %.2f.\n", c_data);
}

__global__ void kernel_2(int N)
{
    int idx = threadIdx.x;
    if (idx < N)
    {

    }   
}

int main(int argc, char **argv)
{ 
    
    int devID = 0;
    hipDeviceProp_t deviceProps;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProps, devID));
    std::cout << "运行GPU设备:" << deviceProps.name << std::endl;

    float h_data = 8.8f;
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_data), &h_data, sizeof(float)));

    dim3 block(1);
    dim3 grid(1);
    kernel_1<<<grid, block>>>();
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpyFromSymbol(&h_data, HIP_SYMBOL(c_data2), sizeof(float)));
    printf("Constant data h_data = %.2f.\n", h_data);

    CUDA_CHECK(hipDeviceReset());

    return 0;
}